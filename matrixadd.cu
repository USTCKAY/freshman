
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, int m, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = 0; i < n; i++)
    for (int j = index; j < m; j += stride)
      x[i * m + j] += y[j];
}

int main(void)
{
  int N = 1<<10;
  int M = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * M * sizeof(float));
  hipMallocManaged(&y, M * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < M; i++)
    y[i] = 2.0f;

  for(int i = 0; i < N; i++)
    for(int j = 0; j < M; j++)
      x[i * M + j] = 1.0f;

  // Run kernel on 1M elements on the GPU
  int blocksize = 256;
  int numblocks = (M + blocksize - 1) / blocksize;
  add<<<numblocks, blocksize>>>(N, M, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    for(int j = 0; j < M; j++)
      maxError = fmax(maxError, fabs(x[i * M + j]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}